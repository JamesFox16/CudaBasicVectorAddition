#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

#define N 512
int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    
    // Allocate space
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Setup values
    random_ints(a, N);
    random_ints(b, N);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with N blocks
    add<<<1,N>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}